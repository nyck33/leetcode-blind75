#include "hip/hip_runtime.h"
// File: ptxTester.cu
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <hip/hip_runtime.h>

extern "C" void computeCovarianceMatrixFromPTX(double* S, double* R, double* Sigma, int sRows, int sCols, int rCols) {
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    hipInit(0);
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, 0);
    hipCtx_t cuContext;
    hipCtxCreate(&cuContext, 0, cuDevice);
    
    // PTX source embedded directly in the code, truncated for brevity
    const char *ptxSource = R"ptx(
        // PTX source should start here, with the entire kernel code as a single string
        .version 6.3
        .target sm_75
        .address_size 64

        .visible .entry _Z14matrixMultiplyPdS_S_iii(
            .param .u64 _Z14matrixMultiplyPdS_S_iii_param_0,
            .param .u64 _Z14matrixMultiplyPdS_S_iii_param_1,
            .param .u64 _Z14matrixMultiplyPdS_S_iii_param_2,
            .param .u32 _Z14matrixMultiplyPdS_S_iii_param_3,
            .param .u32 _Z14matrixMultiplyPdS_S_iii_param_4,
            .param .u32 _Z14matrixMultiplyPdS_S_iii_param_5
        )
        {
            .reg .pred 	%p<9>;
            .reg .b32 	%r<39>;
            .reg .b64 	%rd<30>;
            .reg .f64 	%fd<30>;

            // %bb.0:                               // %entry
            ld.param.u32 	%r20, [_Z14matrixMultiplyPdS_S_iii_param_5];
            ld.param.u32 	%r21, [_Z14matrixMultiplyPdS_S_iii_param_3];
            mov.u32 	%r22, %ctaid.y;
            mov.u32 	%r23, %ntid.y;
            mov.u32 	%r24, %tid.y;
            mad.lo.s32 	%r1, %r22, %r23, %r24;
            mov.u32 	%r25, %ctaid.x;
            mov.u32 	%r26, %ntid.x;
            mov.u32 	%r27, %tid.x;
            mad.lo.s32 	%r2, %r25, %r26, %r27;
            setp.ge.s32 	%p1, %r1, %r21;
            setp.ge.s32 	%p2, %r2, %r20;
            or.pred  	%p3, %p1, %p2;
            @%p3 bra 	$L__BB0_9;
            // %bb.1:                               // %for.cond.preheader
            ld.param.u32 	%r19, [_Z14matrixMultiplyPdS_S_iii_param_4];
            ld.param.u64 	%rd11, [_Z14matrixMultiplyPdS_S_iii_param_2];
            cvta.to.global.u64 	%rd1, %rd11;
            setp.lt.s32 	%p4, %r19, 1;
            mov.f64 	%fd28, 0d0000000000000000;
            @%p4 bra 	$L__BB0_8;
            // %bb.2:                               // %for.body.lr.ph
            ld.param.u64 	%rd10, [_Z14matrixMultiplyPdS_S_iii_param_0];
            ld.param.u64 	%rd12, [_Z14matrixMultiplyPdS_S_iii_param_1];
            cvta.to.global.u64 	%rd2, %rd12;
            cvta.to.global.u64 	%rd3, %rd10;
            mul.lo.s32 	%r3, %r1, %r19;
            and.b32  	%r35, %r19, 3;
            setp.lt.u32 	%p5, %r19, 4;
            mov.f64 	%fd28, 0d0000000000000000;
            mov.u32 	%r34, 0;
            @%p5 bra 	$L__BB0_5;
            // %bb.3:                               // %for.body.lr.ph.new
            and.b32  	%r34, %r19, -4;
            shl.b32 	%r6, %r20, 2;
            shl.b32 	%r7, %r20, 1;
            mul.lo.s32 	%r8, %r20, 3;
            mul.wide.s32 	%rd13, %r3, 8;
            add.s64 	%rd14, %rd13, %rd3;
            add.s64 	%rd29, %rd14, 16;
            mov.f64 	%fd28, 0d0000000000000000;
            mov.u32 	%r37, %r2;
            mov.u32 	%r38, %r34;
            $L__BB0_4:                              // %for.body
                                                    // =>This Inner Loop Header: Depth=1
            ld.global.f64 	%fd12, [%rd29+-16];
            mul.wide.s32 	%rd15, %r37, 8;
            add.s64 	%rd16, %rd2, %rd15;
            ld.global.f64 	%fd13, [%rd16];
            fma.rn.f64 	%fd14, %fd12, %fd13, %fd28;
            ld.global.f64 	%fd15, [%rd29+-8];
            add.s32 	%r29, %r20, %r37;
            mul.wide.s32 	%rd17, %r29, 8;
            add.s64 	%rd18, %rd2, %rd17;
            ld.global.f64 	%fd16, [%rd18];
            fma.rn.f64 	%fd17, %fd15, %fd16, %fd14;
            ld.global.f64 	%fd18, [%rd29];
            add.s32 	%r30, %r7, %r37;
            mul.wide.s32 	%rd19, %r30, 8;
            add.s64 	%rd20, %rd2, %rd19;
            ld.global.f64 	%fd19, [%rd20];
            fma.rn.f64 	%fd20, %fd18, %fd19, %fd17;
            ld.global.f64 	%fd21, [%rd29+8];
            add.s32 	%r31, %r8, %r37;
            mul.wide.s32 	%rd21, %r31, 8;
            add.s64 	%rd22, %rd2, %rd21;
            ld.global.f64 	%fd22, [%rd22];
            fma.rn.f64 	%fd28, %fd21, %fd22, %fd20;
            add.s32 	%r38, %r38, -4;
            add.s32 	%r37, %r37, %r6;
            add.s64 	%rd29, %rd29, 32;
            setp.eq.s32 	%p6, %r38, 0;
            @%p6 bra 	$L__BB0_5;
            bra.uni 	$L__BB0_4;
            $L__BB0_5:                              // %for.cond.cleanup.loopexit.unr-lcssa
            setp.eq.s32 	%p7, %r35, 0;
            @%p7 bra 	$L__BB0_8;
            // %bb.6:                               // %for.body.epil.preheader
            mad.lo.s32 	%r36, %r34, %r20, %r2;
            add.s32 	%r32, %r34, %r3;
            mul.wide.s32 	%rd23, %r32, 8;
            add.s64 	%rd28, %rd3, %rd23;
            $L__BB0_7:                              // %for.body.epil
                                                    // =>This Inner Loop Header: Depth=1
            .pragma "nounroll";
            ld.global.f64 	%fd23, [%rd28];
            mul.wide.s32 	%rd24, %r36, 8;
            add.s64 	%rd25, %rd2, %rd24;
            ld.global.f64 	%fd24, [%rd25];
            fma.rn.f64 	%fd28, %fd23, %fd24, %fd28;
            add.s32 	%r36, %r36, %r20;
            add.s64 	%rd28, %rd28, 8;
            add.s32 	%r35, %r35, -1;
            setp.ne.s32 	%p8, %r35, 0;
            @%p8 bra 	$L__BB0_7;
            $L__BB0_8:                              // %for.cond.cleanup
            mad.lo.s32 	%r33, %r1, %r20, %r2;
            mul.wide.s32 	%rd26, %r33, 8;
            add.s64 	%rd27, %rd1, %rd26;
            st.global.f64 	[%rd27], %fd28;
            $L__BB0_9:                              // %if.end
            ret;
                                                    // -- End function
        }
    )ptx";

    // Load the PTX code into a module
    hipModuleLoadData(&cuModule, ptxSource);
    // Obtain a handle to the kernel function
    hipModuleGetFunction(&cuFunction, cuModule, "_Z14matrixMultiplyPdS_S_iii");

    void* args[] = { &S, &R, &Sigma, &sRows, &sCols, &rCols };

    // Define grid and block dimensions based on the original CUDA code logic
    int threadsPerBlock = 256; // Number of threads per block
    int blockSize = 16; // Block size for x and y dimensions
    int numBlocksX = (sCols + blockSize - 1) / blockSize;
    int numBlocksY = (sRows + blockSize - 1) / blockSize;

    // Launch the kernel
    hipModuleLaunchKernel(cuFunction,
                   numBlocksX, numBlocksY, 1, // Adjusted Grid dimensions
                   blockSize, blockSize, 1, // Block dimensions
                   0, NULL, // Shared memory and stream
                   args, NULL); // Kernel arguments and extra options

    // Synchronize to wait for kernel completion
    hipCtxSynchronize();
    // Clean up
    hipCtxDestroy(cuContext);
}
